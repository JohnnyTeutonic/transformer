#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/cublas_check.cuh"

// Global cuBLAS handle
hipblasHandle_t cublas_handle;

void initialize_cuda() {
    // Select first available GPU
    CUDA_CHECK(hipSetDevice(0));
    
    // Create cuBLAS handle
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    
    // Set cuBLAS to use tensor cores if available
    CUBLAS_CHECK(hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH));
    
    // Enable asynchronous execution
    CUDA_CHECK(hipSetDeviceFlags(hipDeviceScheduleAuto));
}

void cleanup_cuda() {
    // Destroy cuBLAS handle
    if (cublas_handle != nullptr) {
        CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    }
    
    // Reset device
    CUDA_CHECK(hipDeviceReset());
} 