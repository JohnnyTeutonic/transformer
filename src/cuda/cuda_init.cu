#include "../../include/cuda/cuda_check.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <string>

hipblasHandle_t cublas_handle;

void initialize_cuda() {
    // Get number of devices
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        throw std::runtime_error("Failed to get CUDA device count: " +
                                 std::string(hipGetErrorString(error)));
    }

    if (deviceCount == 0) {
        throw std::runtime_error("No CUDA-capable devices found");
    }

    // Get device properties
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0)); // Use first device

    // Print device info
    printf("Using CUDA Device %d: %s\n", 0, deviceProp.name);

    // Set device
    CUDA_CHECK(hipSetDevice(0));

    // Initialize cuBLAS
    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("Failed to initialize cuBLAS");
    }

    // Ensure device is ready
    CUDA_CHECK(hipDeviceSynchronize());
}

void cleanup_cuda() {
    if (cublas_handle != nullptr) {
        hipblasDestroy(cublas_handle);
    }
    hipDeviceReset();
}