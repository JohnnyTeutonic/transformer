#include "../../include/cuda/cuda_check.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/cuda/cuda_init.cuh"
#include "../../include/cuda/cuda_utils.cuh"
#include <stdexcept>
#include <string>

// Move handle into cuda namespace
namespace cuda {
    hipblasHandle_t cublas_handle = nullptr;
}

// Move this inside namespace cuda
namespace cuda {
    static bool cuda_initialized = false;

    void initialize_cuda() {
        if (cuda_initialized) {
            return;  // Already initialized
        }

        try {
            // Set device
            CUDA_CHECK(hipSetDevice(0));

            // Initialize cuBLAS
            CUBLAS_CHECK(hipblasCreate(&cublas_handle));
            
            // Set stream
            hipStream_t stream;
            CUDA_CHECK(hipStreamCreate(&stream));
            CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));

            cuda_initialized = true;
        } catch (const std::exception& e) {
            cleanup_cuda();  // Clean up on failure
            throw std::runtime_error(std::string("CUDA initialization failed: ") + e.what());
        }
    }

    void cleanup_cuda() {
        if (!cuda_initialized) {
            return;  // Already cleaned up
        }

        if (cublas_handle != nullptr) {
            hipblasDestroy(cublas_handle);
            cublas_handle = nullptr;
        }

        hipDeviceReset();
        cuda_initialized = false;
    }

    bool is_initialized() {
        return cuda_initialized;
    }
} // namespace cuda

// Keep these outside namespace
bool init_cublas() {
    hipblasStatus_t status = hipblasCreate(&cuda::cublas_handle);  // Use namespace qualified handle
    return (status == HIPBLAS_STATUS_SUCCESS);
}

void cleanup_cublas() {
    if (cuda::cublas_handle != nullptr) {  // Use namespace qualified handle
        hipblasDestroy(cuda::cublas_handle);
        cuda::cublas_handle = nullptr;
    }
}

// Add any other CUDA initialization code here...

// When shutting down
void shutdown() {
    cleanup_cublas();
    cleanup_cuda();  // Now this should be found since we included cuda_utils.cuh
}