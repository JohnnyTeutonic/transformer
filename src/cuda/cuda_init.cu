#include "../../include/cuda/cuda_check.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/cuda/cuda_init.cuh"
#include "../../include/cuda/cuda_utils.cuh"
#include <stdexcept>
#include <string>

namespace cuda {
    hipblasHandle_t cublas_handle = nullptr;
    static bool cuda_initialized = false;

    bool is_initialized() {
        return cuda_initialized;
    }

    void initialize_cuda() {
        if (cuda_initialized) {
            return;  // Already initialized
        }

        try {
            // Set device
            CUDA_CHECK(hipSetDevice(0));

            // Initialize cuBLAS
            CUBLAS_CHECK(hipblasCreate(&cublas_handle));
            
            // Set stream
            hipStream_t stream;
            CUDA_CHECK(hipStreamCreate(&stream));
            CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));

            cuda_initialized = true;
        } catch (const std::exception& e) {
            cleanup_cuda();  // Clean up on failure
            throw std::runtime_error(std::string("CUDA initialization failed: ") + e.what());
        }
    }

    void cleanup_cuda() {
        if (!cuda_initialized) {
            return;  // Already cleaned up
        }

        if (cublas_handle != nullptr) {
            hipblasDestroy(cublas_handle);
            cublas_handle = nullptr;
        }

        hipDeviceReset();
        cuda_initialized = false;
    }
} // namespace cuda

// Keep these outside namespace
bool init_cublas() {
    hipblasStatus_t status = hipblasCreate(&cuda::cublas_handle);  // Use namespace qualified handle
    return (status == HIPBLAS_STATUS_SUCCESS);
}

void cleanup_cublas() {
    if (cuda::cublas_handle != nullptr) {  // Use namespace qualified handle
        hipblasDestroy(cuda::cublas_handle);
        cuda::cublas_handle = nullptr;
    }
}

// Add any other CUDA initialization code here...

// When shutting down
void shutdown() {
    cuda::cleanup_cuda();  // This handles both CUDA and cuBLAS cleanup
}