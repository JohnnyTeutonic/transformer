#include "../../include/cuda/cuda_check.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/cuda/cuda_init.cuh"
#include <stdexcept>
#include <string>

// Global cuBLAS handle
hipblasHandle_t cublas_handle = nullptr;
static bool cuda_initialized = false;

namespace cuda {

void initialize_cuda() {
    if (cuda_initialized) {
        return;  // Already initialized
    }

    try {
        // Set device
        CUDA_CHECK(hipSetDevice(0));

        // Initialize cuBLAS
        CUBLAS_CHECK(hipblasCreate(&cublas_handle));
        
        // Set stream
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));
        CUBLAS_CHECK(hipblasSetStream(cublas_handle, stream));

        cuda_initialized = true;
    } catch (const std::exception& e) {
        cleanup_cuda();  // Clean up on failure
        throw std::runtime_error(std::string("CUDA initialization failed: ") + e.what());
    }
}

void cleanup_cuda() {
    if (!cuda_initialized) {
        return;  // Already cleaned up
    }

    if (cublas_handle != nullptr) {
        hipblasDestroy(cublas_handle);
        cublas_handle = nullptr;
    }

    hipDeviceReset();
    cuda_initialized = false;
}

bool is_initialized() {
    return cuda_initialized;
}

} // namespace cuda