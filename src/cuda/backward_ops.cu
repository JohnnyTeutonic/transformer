#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/cuda/backward_ops.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/cuda_utils.cuh"

namespace cuda {
    // Forward declare kernels
    __global__ void layer_norm_backward_kernel(const float* grad, const float* input,
                                             const float* gamma, float* dx, int batch_size,
                                             int hidden_size, float eps);
    __global__ void gelu_backward_kernel(float* grad_output, const float* input, int size);

    void layer_norm_backward(const Matrix& grad, const Matrix& input, const Matrix& gamma,
                           Matrix& dx, float eps) {
        size_t batch_size = input.rows();
        size_t hidden_size = input.cols();
        
        size_t grad_size = grad.size() * sizeof(float);
        size_t input_size = input.size() * sizeof(float);
        size_t gamma_size = gamma.size() * sizeof(float);

        float *d_grad, *d_input, *d_gamma, *d_dx;
        
        // Allocate device memory
        CUDA_CHECK(hipMalloc(&d_grad, grad_size));
        CUDA_CHECK(hipMalloc(&d_input, input_size));
        CUDA_CHECK(hipMalloc(&d_gamma, gamma_size));
        CUDA_CHECK(hipMalloc(&d_dx, grad_size));

        // Copy data to device
        CUDA_CHECK(hipMemcpy(d_grad, grad.get_data(), grad_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_input, input.get_data(), input_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_gamma, gamma.get_data(), gamma_size, hipMemcpyHostToDevice));

        // Launch kernel
        dim3 block(256);
        dim3 grid((batch_size * hidden_size + block.x - 1) / block.x);
        
        layer_norm_backward_kernel<<<grid, block>>>(d_grad, d_input, d_gamma, d_dx,
                                                  batch_size, hidden_size, eps);

        // Copy result back to host
        CUDA_CHECK(hipMemcpy(dx.get_data(), d_dx, grad_size, hipMemcpyDeviceToHost));

        // Free device memory
        CUDA_CHECK(hipFree(d_grad));
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_gamma));
        CUDA_CHECK(hipFree(d_dx));
    }
    
    void gelu_backward(Matrix& grad_output, const Matrix& input) {
        float *d_grad, *d_input;
        size_t size = input.size() * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_grad, size));
        CUDA_CHECK(hipMalloc(&d_input, size));

        CUDA_CHECK(hipMemcpy(d_grad, grad_output.get_data(), size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_input, input.get_data(), size, hipMemcpyHostToDevice));

        dim3 block(256);
        dim3 grid((input.size() + block.x - 1) / block.x);

        gelu_backward_kernel<<<grid, block>>>(d_grad, d_input, input.size());

        CUDA_CHECK(hipMemcpy(grad_output.get_data(), d_grad, size, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_grad));
        CUDA_CHECK(hipFree(d_input));
    }

    __global__ void layer_norm_backward_kernel(const float* grad, const float* input,
                                             const float* gamma, float* dx, int batch_size,
                                             int hidden_size, float eps) {
        extern __shared__ float shared_mem[];
        float* mean = shared_mem;
        float* var = shared_mem + blockDim.x;
        float* sum_grad = shared_mem + 2 * blockDim.x;
        float* sum_grad_diff = shared_mem + 3 * blockDim.x;

        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < batch_size) {
            // Compute mean and variance
            float batch_mean = 0.0f;
            float batch_var = 0.0f;
            
            for (int j = 0; j < hidden_size; ++j) {
                batch_mean += input[tid * hidden_size + j];
            }
            batch_mean /= hidden_size;
            
            for (int j = 0; j < hidden_size; ++j) {
                float diff = input[tid * hidden_size + j] - batch_mean;
                batch_var += diff * diff;
            }
            batch_var /= hidden_size;
            
            float std = sqrtf(batch_var + eps);
            
            // Compute gradients
            float batch_sum_grad = 0.0f;
            float batch_sum_grad_diff = 0.0f;
            
            for (int j = 0; j < hidden_size; ++j) {
                float diff = input[tid * hidden_size + j] - batch_mean;
                batch_sum_grad += grad[tid * hidden_size + j] * gamma[j];
                batch_sum_grad_diff += grad[tid * hidden_size + j] * gamma[j] * diff;
            }
            
            // Compute final gradients
            for (int j = 0; j < hidden_size; ++j) {
                float diff = input[tid * hidden_size + j] - batch_mean;
                dx[tid * hidden_size + j] = gamma[j] * 
                    (grad[tid * hidden_size + j] - 
                     (batch_sum_grad + diff * batch_sum_grad_diff / batch_var) / hidden_size) / std;
            }
        }
    }

    __global__ void gelu_backward_kernel(float* grad_output, const float* input, int size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            float x = input[idx];
            float cdf = 0.5f * (1.0f + erf(x / sqrtf(2.0f)));
            float pdf = exp(-0.5f * x * x) / sqrtf(2.0f * M_PI);
            grad_output[idx] *= (cdf + x * pdf);
        }
    }
} 