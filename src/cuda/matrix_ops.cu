#include "hip/hip_runtime.h"
#include "../include/cuda/cuda_utils.cuh"

// Global cuBLAS handle definition
hipblasHandle_t cublas_handle;

void initialize_cuda() {
    CUDA_CHECK(hipSetDevice(0));
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
}

void cleanup_cuda() {
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
}

// Optimized matrix multiplication kernel
__global__ void matrix_multiply_kernel(const float* A, const float* B, float* C,
                                     int M, int N, int K) {
    // Use shared memory for better performance
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float sum = 0.0f;
    
    for (int tile = 0; tile < (K + 31) / 32; ++tile) {
        // Load data into shared memory
        if (row < M && tile * 32 + threadIdx.x < K)
            shared_A[threadIdx.y][threadIdx.x] = A[row * K + tile * 32 + threadIdx.x];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;
            
        if (col < N && tile * 32 + threadIdx.y < K)
            shared_B[threadIdx.y][threadIdx.x] = B[(tile * 32 + threadIdx.y) * N + col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;
            
        __syncthreads();
        
        // Compute partial dot product
        if (row < M && col < N) {
            for (int k = 0; k < 32; ++k) {
                sum += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
            }
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
} 