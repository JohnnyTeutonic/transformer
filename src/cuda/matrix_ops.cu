#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/cuda/matrix_ops.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/cuda_utils.cuh"
#include <unordered_map>

// Forward declare all kernels
__global__ void matrix_multiply_kernel(const float* A, const float* B, float* C, 
                                      int M, int N, int K);
__global__ void gelu_forward_kernel(float* x, int size);

namespace cuda {
    // Memory pool for GPU buffers
    struct MemoryPool {
        std::unordered_map<size_t, std::vector<float*>> free_buffers;
        std::unordered_map<float*, size_t> buffer_sizes;
        
        float* allocate(size_t size) {
            // Check if we have a free buffer of the right size
            auto& buffers = free_buffers[size];
            if (!buffers.empty()) {
                float* buffer = buffers.back();
                buffers.pop_back();
                return buffer;
            }
            
            // Allocate new buffer
            float* buffer;
            CUDA_CHECK(hipMalloc(&buffer, size * sizeof(float)));
            buffer_sizes[buffer] = size;
            return buffer;
        }
        
        void free(float* buffer) {
            if (buffer == nullptr) return;
            auto size = buffer_sizes[buffer];
            free_buffers[size].push_back(buffer);
        }
        
        void cleanup() {
            for (auto& pair : free_buffers) {
                for (float* buffer : pair.second) {
                    hipFree(buffer);
                }
            }
            free_buffers.clear();
            buffer_sizes.clear();
        }
    };
    
    static MemoryPool memory_pool;
    // Global cuBLAS handle with proper initialization
    static hipblasHandle_t cublas_handle = nullptr;
    static bool cuda_initialized = false;

    void initialize_cuda() {
        if (cuda_initialized) {
            return;
        }

        // Set CUDA device
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to set CUDA device: " + std::string(hipGetErrorString(err)));
        }
        std::cout << "CUDA device set successfully" << std::endl;

        // Print CUDA device properties
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        std::cout << "Using CUDA device: " << prop.name << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;

        // Initialize cuBLAS
        hipblasStatus_t status = hipblasCreate(&cublas_handle);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle: " + std::to_string(status));
        }
        std::cout << "cuBLAS handle created successfully" << std::endl;

        cuda_initialized = true;
    }

    void cleanup_cuda() {
        if (cublas_handle != nullptr) {
            memory_pool.cleanup();
            hipblasDestroy(cublas_handle);
            cublas_handle = nullptr;
            cuda_initialized = false;
            std::cout << "cuBLAS handle destroyed successfully" << std::endl;
        }
    }

    void matmul(const Matrix& A, const Matrix& B, Matrix& C) {
        // A: [batch_size x hidden_size]
        // B: [hidden_size x vocab_size]
        // C: [batch_size x vocab_size]
        
        // Ensure CUDA is initialized
        if (!cuda_initialized || cublas_handle == nullptr) {
            initialize_cuda();
        }

        // Verify dimensions
        if (A.cols() != B.rows()) {
            throw std::runtime_error("Matrix multiplication dimension mismatch: " +
                std::to_string(A.rows()) + "x" + std::to_string(A.cols()) + " * " +
                std::to_string(B.rows()) + "x" + std::to_string(B.cols()));
        }
        // Ensure output matrix has correct dimensions
        if (C.rows() != A.rows() || C.cols() != B.cols()) {
            throw std::runtime_error("Output matrix has wrong dimensions: expected " +
                std::to_string(A.rows()) + "x" + std::to_string(B.cols()) + " got " +
                std::to_string(C.rows()) + "x" + std::to_string(C.cols()));
        }

        // Use memory pool instead of direct allocation
        float *d_A = memory_pool.allocate(A.rows() * A.cols());
        float *d_B = memory_pool.allocate(B.rows() * B.cols());
        float *d_C = memory_pool.allocate(C.rows() * C.cols());

        size_t A_size = A.rows() * A.cols() * sizeof(float);
        size_t B_size = B.rows() * B.cols() * sizeof(float);
        size_t C_size = C.rows() * C.cols() * sizeof(float);

        // Use asynchronous memory transfers
        hipStream_t stream;
        hipStreamCreate(&stream);
        
        CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), A_size, hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), B_size, hipMemcpyHostToDevice, stream));

        float alpha = 1.0f;
        float beta = 0.0f;

        // Set stream for cuBLAS operation
        hipblasSetStream(cublas_handle, stream);

        hipblasStatus_t status = hipblasSgemm(cublas_handle,
                                          HIPBLAS_OP_N, HIPBLAS_OP_N,
                                          B.cols(), A.rows(), A.cols(),
                                          &alpha,
                                          d_B, B.cols(),
                                          d_A, A.cols(),
                                          &beta,
                                          d_C, B.cols());

        if (status != HIPBLAS_STATUS_SUCCESS) {
            memory_pool.free(d_A);
            memory_pool.free(d_B);
            memory_pool.free(d_C);
            hipStreamDestroy(stream);
            throw std::runtime_error("cuBLAS matrix multiplication failed: " + std::to_string(status));
        }

        CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, C_size, hipMemcpyDeviceToHost, stream));
        
        // Synchronize stream before returning buffers to pool
        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);

        memory_pool.free(d_A);
        memory_pool.free(d_B);
        memory_pool.free(d_C);
    }

    void matmul_transposed(const Matrix& A, const Matrix& B, Matrix& C) {
        // Ensure CUDA is initialized
        if (!cuda_initialized || cublas_handle == nullptr) {
            initialize_cuda();
        }

        // Verify dimensions for transposed multiplication
        // A: [M x K], B: [N x K] (transposed), C: [M x N]
        if (A.cols() != B.cols()) {
            throw std::runtime_error("Matrix multiplication dimension mismatch for transposed operation: " +
                std::to_string(A.rows()) + "x" + std::to_string(A.cols()) + " * " +
                std::to_string(B.rows()) + "x" + std::to_string(B.cols()));
        }
        
        // Ensure output matrix has correct dimensions
        if (C.rows() != A.rows() || C.cols() != B.rows()) {
            throw std::runtime_error("Output matrix has wrong dimensions: expected " +
                std::to_string(A.rows()) + "x" + std::to_string(B.rows()) + " got " +
                std::to_string(C.rows()) + "x" + std::to_string(C.cols()));
        }

        float* d_A, *d_B, *d_C;
        size_t A_size = A.rows() * A.cols() * sizeof(float);
        size_t B_size = B.rows() * B.cols() * sizeof(float);
        size_t C_size = C.rows() * C.cols() * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_A, A_size));
        CUDA_CHECK(hipMalloc(&d_B, B_size));
        CUDA_CHECK(hipMalloc(&d_C, C_size));

        CUDA_CHECK(hipMemcpy(d_A, A.data(), A_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B, B.data(), B_size, hipMemcpyHostToDevice));

        float alpha = 1.0f;
        float beta = 0.0f;

        // For A[M,K] * B[N,K]^T = C[M,N], we compute:
        // C = A * B^T
        hipblasStatus_t status = hipblasSgemm(cublas_handle,
                                          HIPBLAS_OP_T, HIPBLAS_OP_N,  // Transpose B, no transpose A
                                          B.rows(), A.rows(), A.cols(),  // M, N, K dimensions
                                          &alpha,
                                          d_B, B.cols(),  // Leading dimension is cols for B
                                          d_A, A.cols(),  // Leading dimension is cols for A
                                          &beta,
                                          d_C, B.rows()); // Leading dimension is rows of B for C

        if (status != HIPBLAS_STATUS_SUCCESS) {
            CUDA_CHECK(hipFree(d_A));
            CUDA_CHECK(hipFree(d_B));
            CUDA_CHECK(hipFree(d_C));
            throw std::runtime_error("cuBLAS matrix multiplication failed: " + std::to_string(status));
        }

        CUDA_CHECK(hipMemcpy(C.data(), d_C, C_size, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_B));
        CUDA_CHECK(hipFree(d_C));
    }

    void gelu_forward(Matrix& x) {
        float* d_x;
        size_t size = x.size() * sizeof(float);
        
        CUDA_CHECK(hipMalloc(&d_x, size));
        CUDA_CHECK(hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice));
        
        dim3 block(256);
        dim3 grid((x.size() + 255) / 256);
        
        gelu_forward_kernel<<<grid, block>>>(d_x, x.size());
        
        CUDA_CHECK(hipMemcpy(x.data(), d_x, size, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_x));
    }
}

// Kernel implementations
__global__ void matrix_multiply_kernel(const float* A, const float* B, float* C, 
                                     int M, int N, int K) {
    // Use shared memory for better performance
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    for (int tile = 0; tile < (K + 31) / 32; ++tile) {
        // Load data into shared memory
        if (row < M && tile * 32 + threadIdx.x < K)
            shared_A[threadIdx.y][threadIdx.x] = A[row * K + tile * 32 + threadIdx.x];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && tile * 32 + threadIdx.y < K)
            shared_B[threadIdx.y][threadIdx.x] = B[(tile * 32 + threadIdx.y) * N + col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Compute partial dot product
        if (row < M && col < N) {
            for (int k = 0; k < 32; ++k) {
                sum += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
            }
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// GELU kernel implementations
__global__ void gelu_forward_kernel(float* x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = x[idx];
        float cdf = 0.5f * (1.0f + tanhf(0.797884f * (val + 0.044715f * val * val * val)));
        x[idx] = val * cdf;
    }
}