#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/cuda/attention_ops.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/kernel_declarations.cuh"

namespace cuda {
    // Forward declare kernels
    CUDA_KERNEL void attention_scores_kernel(const float* Q, const float* K, float* scores,
                                                      float scale, int seq_len, int head_dim);
    CUDA_KERNEL void softmax_kernel(float* matrix, int rows, int cols);
    CUDA_KERNEL void attention_kernel(const float* Q, const float* K, const float* V,
                                   float* output, int batch_size, int seq_len, int head_dim);

    void compute_attention_scores(const Matrix& Q, const Matrix& K, Matrix& scores, float scale, int num_heads) {
        // Q and K are [batch_size x hidden_dim]
        int batch_size = Q.rows();
        int hidden_dim = Q.cols();
        int head_dim = hidden_dim / num_heads;
        int seq_len = batch_size;  // In this case, seq_len is same as batch_size
        
        // Verify scores dimensions
        if (scores.rows() != batch_size || scores.cols() != seq_len) {
            throw std::runtime_error("Scores matrix has wrong dimensions: expected " +
                std::to_string(batch_size) + "x" + std::to_string(seq_len) + " got " +
                std::to_string(scores.rows()) + "x" + std::to_string(scores.cols()));
        }

        // Launch kernel with correct dimensions
        dim3 block(32, 32);
        dim3 grid((seq_len + 31) / 32, (seq_len + 31) / 32);
        
        float* d_Q, *d_K, *d_scores;
        size_t Q_size = Q.size() * sizeof(float);
        size_t K_size = K.size() * sizeof(float);
        size_t scores_size = scores.size() * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_Q, Q_size));
        CUDA_CHECK(hipMalloc(&d_K, K_size));
        CUDA_CHECK(hipMalloc(&d_scores, scores_size));

        CUDA_CHECK(hipMemcpy(d_Q, Q.data(), Q_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_K, K.data(), K_size, hipMemcpyHostToDevice));
        
        attention_scores_kernel<<<grid, block>>>(d_Q, d_K, d_scores,
            scale, seq_len, head_dim);

        CUDA_CHECK(hipMemcpy(scores.data(), d_scores, scores_size, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_Q));
        CUDA_CHECK(hipFree(d_K));
        CUDA_CHECK(hipFree(d_scores));
    }

    void apply_softmax(Matrix& matrix) {
        float* d_matrix;
        size_t size = matrix.size() * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_matrix, size));
        CUDA_CHECK(hipMemcpy(d_matrix, matrix.data(), size, hipMemcpyHostToDevice));

        softmax_kernel<<<matrix.rows(), 1>>>(d_matrix, matrix.rows(), matrix.cols());

        CUDA_CHECK(hipMemcpy(matrix.data(), d_matrix, size, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_matrix));
    }

    void attention_forward(const Matrix& Q, const Matrix& K, const Matrix& V, 
                         Matrix& output, int batch_size, int num_heads, int seq_len) {
        dim3 grid(batch_size, num_heads);
        
        float *d_Q, *d_K, *d_V, *d_output;
        size_t QKV_size = Q.size() * sizeof(float);
        size_t output_size = output.size() * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_Q, QKV_size));
        CUDA_CHECK(hipMalloc(&d_K, QKV_size));
        CUDA_CHECK(hipMalloc(&d_V, QKV_size));
        CUDA_CHECK(hipMalloc(&d_output, output_size));

        CUDA_CHECK(hipMemcpy(d_Q, Q.data(), QKV_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_K, K.data(), QKV_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_V, V.data(), QKV_size, hipMemcpyHostToDevice));

        attention_kernel<<<grid, seq_len>>>(d_Q, d_K, d_V, d_output, 
                                          batch_size, seq_len, Q.cols() / num_heads);

        CUDA_CHECK(hipMemcpy(output.data(), d_output, output_size, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_Q));
        CUDA_CHECK(hipFree(d_K));
        CUDA_CHECK(hipFree(d_V));
        CUDA_CHECK(hipFree(d_output));
    }

    // CUDA kernel launcher
    void launch_attention_scores_kernel(const float* Q, const float* K, float* scores, float scale,
                                      int seq_len, int head_dim, hipStream_t stream) {
        dim3 block_dim(16, 16);
        dim3 grid_dim((seq_len + block_dim.x - 1) / block_dim.x,
                      (seq_len + block_dim.y - 1) / block_dim.y);

        attention_scores_kernel<<<grid_dim, block_dim, 0, stream>>>(Q, K, scores, scale, seq_len,
                                                                 head_dim);
    }
}

// Kernel implementations
CUDA_KERNEL void attention_scores_kernel(const float* Q, const float* K, float* scores,
                                                    float scale, int seq_len, int head_dim) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < seq_len && col < seq_len) {
        float sum = 0.0f;
        for (int i = 0; i < head_dim; i++) {
            sum += Q[row * head_dim + i] * K[col * head_dim + i];
        }
        scores[row * seq_len + col] = sum * scale;
    }
}

CUDA_KERNEL void softmax_kernel(float* matrix, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        // Find max for numerical stability
        float max_val = matrix[row * cols];
        for (int i = 1; i < cols; i++) {
            max_val = max(max_val, matrix[row * cols + i]);
        }

        // Compute exp and sum
        float sum = 0.0f;
        for (int i = 0; i < cols; i++) {
            matrix[row * cols + i] = expf(matrix[row * cols + i] - max_val);
            sum += matrix[row * cols + i];
        }

        // Normalize
        for (int i = 0; i < cols; i++) {
            matrix[row * cols + i] /= sum;
        }
    }
}

CUDA_KERNEL void attention_kernel(const float* Q, const float* K, const float* V,
                               float* output, int batch_size, int seq_len, int head_dim) {
    int b = blockIdx.x;  // batch index
    int h = blockIdx.y;  // head index
    int i = threadIdx.x; // sequence position

    if (i < seq_len) {
        // Compute attention scores
        float scores[1024];  // Assuming max sequence length
        for (int j = 0; j < seq_len; j++) {
            float score = 0.0f;
            for (int d = 0; d < head_dim; d++) {
                score += Q[b * seq_len * head_dim + i * head_dim + d] *
                        K[b * seq_len * head_dim + j * head_dim + d];
            }
            scores[j] = score / sqrtf(float(head_dim));
        }

        // Apply softmax
        float max_score = scores[0];
        for (int j = 1; j < seq_len; j++) {
            max_score = max(max_score, scores[j]);
        }

        float sum = 0.0f;
        for (int j = 0; j < seq_len; j++) {
            scores[j] = expf(scores[j] - max_score);
            sum += scores[j];
        }

        for (int j = 0; j < seq_len; j++) {
            scores[j] /= sum;
        }

        // Compute weighted sum
        for (int d = 0; d < head_dim; d++) {
            float weighted_sum = 0.0f;
            for (int j = 0; j < seq_len; j++) {
                weighted_sum += scores[j] * V[b * seq_len * head_dim + j * head_dim + d];
            }
            output[b * seq_len * head_dim + i * head_dim + d] = weighted_sum;
        }
    }
} 