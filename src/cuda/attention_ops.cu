#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/attention.hpp"
#include "../../include/cuda/attention_ops.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/matrix_ops.cuh"
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/kernel_declarations.cuh"

#ifndef MAX_SEQ_LEN
#define MAX_SEQ_LEN 2048  // or whatever maximum sequence length you want to support
#endif

// Kernel declarations in extern "C" to match header
extern "C" {
    __global__ void attention_kernel(const float* Q, const float* K, const float* V,
                                   float* output, const float* mask,
                                   int batch_size, int seq_len, int head_dim, int hidden_dim);
    
    __global__ void attention_scores_kernel(const float* Q, const float* K, float* scores,
                                          float scale, int seq_len, int head_dim);
    
    __global__ void softmax_kernel(float* matrix, int rows, int cols);
}

namespace cuda {
    // Host functions only in namespace
    void compute_attention_scores(const Matrix& Q, const Matrix& K, Matrix& scores, float scale, int num_heads) {
        // Synchronize before starting
        CUDA_CHECK(hipDeviceSynchronize());
        
        int batch_size = Q.rows();
        int hidden_dim = Q.cols();
        int head_dim = hidden_dim / num_heads;
        int seq_len = batch_size;

        // Verify all dimensions are valid
        if (batch_size <= 0 || hidden_dim <= 0 || head_dim <= 0 || seq_len <= 0) {
            throw std::runtime_error("Invalid dimensions detected");
        }

        // Memory allocation with error checking
        float* d_Q = nullptr;
        float* d_K = nullptr;
        float* d_scores = nullptr;
        
        try {
            CUDA_CHECK(hipMalloc(&d_Q, Q.size() * sizeof(float)));
            CUDA_CHECK(hipMalloc(&d_K, K.size() * sizeof(float)));
            CUDA_CHECK(hipMalloc(&d_scores, scores.size() * sizeof(float)));
            
            // Zero initialize the scores buffer
            CUDA_CHECK(hipMemset(d_scores, 0, scores.size() * sizeof(float)));

            CUDA_CHECK(hipMemcpy(d_Q, Q.data(), Q.size() * sizeof(float), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpy(d_K, K.data(), K.size() * sizeof(float), hipMemcpyHostToDevice));
            
            // Synchronize to ensure memory transfers are complete
            CUDA_CHECK(hipDeviceSynchronize());

            dim3 block(16, 16);
            dim3 grid((seq_len + block.x - 1) / block.x, (seq_len + block.y - 1) / block.y);
            
            attention_scores_kernel<<<grid, block>>>(d_Q, d_K, d_scores,
                scale, seq_len, head_dim);
                
            // Check for kernel launch errors
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                throw std::runtime_error(std::string("Kernel launch failed: ") + 
                                       hipGetErrorString(err));
            }

            // Synchronize after kernel
            CUDA_CHECK(hipDeviceSynchronize());

            CUDA_CHECK(hipMemcpy(scores.data(), d_scores, scores.size() * sizeof(float), 
                                hipMemcpyDeviceToHost));

        } catch (const std::exception& e) {
            printf("CUDA error caught: %s\n", e.what());
            // Clean up on error
            if (d_Q) hipFree(d_Q);
            if (d_K) hipFree(d_K);
            if (d_scores) hipFree(d_scores);
            throw;  // Re-throw the exception
        }

        // Clean up
        CUDA_CHECK(hipFree(d_Q));
        CUDA_CHECK(hipFree(d_K));
        CUDA_CHECK(hipFree(d_scores));
        
        // Final synchronize
        CUDA_CHECK(hipDeviceSynchronize());
    }

    void apply_softmax(Matrix& matrix) {
        float* d_matrix;
        size_t size = matrix.size() * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_matrix, size));
        CUDA_CHECK(hipMemcpy(d_matrix, matrix.data(), size, hipMemcpyHostToDevice));

        softmax_kernel<<<matrix.rows(), 1>>>(d_matrix, matrix.rows(), matrix.cols());

        CUDA_CHECK(hipMemcpy(matrix.data(), d_matrix, size, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_matrix));
    }

    void attention_forward(const Matrix& Q, const Matrix& K, const Matrix& V, 
                         Matrix& output, int batch_size, int num_heads, int seq_len) {
        // Configure grid for batch and head parallelism
        dim3 block(32, 1);
        dim3 grid((batch_size + block.x - 1) / block.x, num_heads);
        
        int head_dim = Q.cols() / num_heads;
        int hidden_dim = Q.cols();  // Store the full hidden dimension

        float *d_Q, *d_K, *d_V, *d_output;
        size_t QKV_size = Q.size() * sizeof(float);
        size_t output_size = output.size() * sizeof(float);
        CUDA_CHECK(hipMalloc(&d_Q, QKV_size));
        CUDA_CHECK(hipMalloc(&d_K, QKV_size));
        CUDA_CHECK(hipMalloc(&d_V, QKV_size));
        CUDA_CHECK(hipMalloc(&d_output, output_size));

        CUDA_CHECK(hipMemcpy(d_Q, Q.data(), QKV_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_K, K.data(), QKV_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_V, V.data(), QKV_size, hipMemcpyHostToDevice));

        // Allocate shared memory for scores
        size_t shared_mem_size = seq_len * sizeof(float);
        attention_kernel<<<grid, block, shared_mem_size>>>(d_Q, d_K, d_V, d_output,
                                                         batch_size, seq_len, head_dim, hidden_dim);

        CUDA_CHECK(hipMemcpy(output.data(), d_output, output_size, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(d_Q));
        CUDA_CHECK(hipFree(d_K));
        CUDA_CHECK(hipFree(d_V));
        CUDA_CHECK(hipFree(d_output));
    }

    void launch_attention_scores_kernel(const float* Q, const float* K, float* scores, float scale,
                                      int seq_len, int head_dim, hipStream_t stream) {
        dim3 block_dim(16, 16);
        dim3 grid_dim((seq_len + block_dim.x - 1) / block_dim.x,
                      (seq_len + block_dim.y - 1) / block_dim.y);

        attention_scores_kernel<<<grid_dim, block_dim, 0, stream>>>(Q, K, scores, scale, seq_len,
                                                                 head_dim);
    }

    void launch_attention_kernel(const float* Q, const float* K, const float* V,
                               float* output, const float* mask,
                               int batch_size, int num_heads, int seq_len, int head_dim,
                               float scale, hipStream_t stream) {
        // Calculate grid and block dimensions
        dim3 block(256);
        dim3 grid((seq_len + block.x - 1) / block.x, num_heads, batch_size);
        
        // Launch kernel
        scaled_dot_product_attention_kernel<<<grid, block, 0, stream>>>(
            Q, K, V, output, mask,
            batch_size, num_heads, seq_len, head_dim,
            scale);
        
        CUDA_CHECK(hipGetLastError());
    }
}

// Kernel implementations
extern "C" {
    __global__ void attention_kernel(const float* Q, const float* K, const float* V,
                                   float* output, const float* mask,
                                   int batch_size, int seq_len, int head_dim, int hidden_dim) {
        int b = blockIdx.x * blockDim.x + threadIdx.x;  // batch index
        int h = blockIdx.y;  // head index

        if (b < batch_size) {
            // Process this batch element for the current head
            int head_offset = h * head_dim;
            int batch_offset = b * hidden_dim;
            
            // Allocate scores in shared memory
            extern __shared__ float scores[];
            
            // Compute attention scores for this head
            for (int j = 0; j < seq_len; j++) {
                float score = 0.0f;
                for (int d = 0; d < head_dim; d++) {
                    int q_idx = batch_offset + head_offset + d;
                    int k_idx = j * hidden_dim + head_offset + d;
                    
                    if (q_idx < batch_size * hidden_dim && k_idx < seq_len * hidden_dim) {
                        score += Q[q_idx] * K[k_idx];
                    }
                }
                scores[j] = score / sqrtf(float(head_dim));
                
                // Apply mask if provided
                if (mask != nullptr) {
                    scores[j] += mask[b * seq_len + j];
                }
            }
            
            // Apply softmax
            float max_score = scores[0];
            for (int j = 1; j < seq_len; j++) {
                max_score = max(max_score, scores[j]);
            }
            
            float sum = 0.0f;
            for (int j = 0; j < seq_len; j++) {
                scores[j] = expf(scores[j] - max_score);
                sum += scores[j];
            }
            
            for (int j = 0; j < seq_len; j++) {
                scores[j] /= sum;
            }
            
            // Compute weighted sum
            for (int d = 0; d < head_dim; d++) {
                float weighted_sum = 0.0f;
                for (int j = 0; j < seq_len; j++) {
                    int v_idx = j * hidden_dim + head_offset + d;
                    if (v_idx < seq_len * hidden_dim) {
                        weighted_sum += scores[j] * V[v_idx];
                    }
                }
                int out_idx = batch_offset + head_offset + d;
                if (out_idx < batch_size * hidden_dim) {
                    output[out_idx] = weighted_sum;
                }
            }
        }
    }

    __global__ void attention_scores_kernel(const float* Q, const float* K, float* scores,
                                           float scale, int seq_len, int head_dim) {
        int row = blockIdx.x * blockDim.x + threadIdx.x;
        int col = blockIdx.y * blockDim.y + threadIdx.y;

        if (row < seq_len && col < seq_len) {
            float sum = 0.0f;
            for (int i = 0; i < head_dim; i++) {
                int q_idx = row * head_dim + i;
                int k_idx = col * head_dim + i;
                if (q_idx < seq_len * head_dim && k_idx < seq_len * head_dim) {
                    sum += Q[q_idx] * K[k_idx];
                }
            }
            scores[row * seq_len + col] = sum * scale;
        }
    }

    __global__ void softmax_kernel(float* matrix, int rows, int cols) {
        int row = blockIdx.x * blockDim.x + threadIdx.x;
        if (row < rows) {
            // Find max for numerical stability
            float max_val = matrix[row * cols];
            for (int i = 1; i < cols; i++) {
                max_val = max(max_val, matrix[row * cols + i]);
            }

            // Compute exp and sum
            float sum = 0.0f;
            for (int i = 0; i < cols; i++) {
                matrix[row * cols + i] = expf(matrix[row * cols + i] - max_val);
                sum += matrix[row * cols + i];
            }

            // Normalize
            for (int i = 0; i < cols; i++) {
                matrix[row * cols + i] /= (sum + 1e-6f);
            }
        }
    }

    __global__ void scaled_dot_product_attention_kernel(
        const float* Q, const float* K, const float* V,
        float* output, const float* mask,
        int batch_size, int num_heads, int seq_len, int head_dim,
        float scale) {
        
        const int b = blockIdx.z;  // batch index
        const int h = blockIdx.y;  // head index
        const int i = blockIdx.x * blockDim.x + threadIdx.x;  // sequence position
        
        if (b < batch_size && h < num_heads && i < seq_len) {
            // Calculate attention scores for this position
            float scores[MAX_SEQ_LEN];  // Assume MAX_SEQ_LEN is defined
            
            // Compute attention scores
            for (int j = 0; j < seq_len; j++) {
                float sum = 0.0f;
                for (int d = 0; d < head_dim; d++) {
                    const int q_idx = ((b * num_heads + h) * seq_len + i) * head_dim + d;
                    const int k_idx = ((b * num_heads + h) * seq_len + j) * head_dim + d;
                    sum += Q[q_idx] * K[k_idx];
                }
                scores[j] = sum * scale;
                
                // Apply mask if provided
                if (mask != nullptr) {
                    scores[j] += mask[i * seq_len + j];
                }
            }
            
            // Apply softmax
            float max_score = scores[0];
            for (int j = 1; j < seq_len; j++) {
                max_score = max(max_score, scores[j]);
            }
            
            float sum = 0.0f;
            for (int j = 0; j < seq_len; j++) {
                scores[j] = expf(scores[j] - max_score);
                sum += scores[j];
            }
            
            for (int j = 0; j < seq_len; j++) {
                scores[j] /= sum;
            }
            
            // Compute weighted sum of values
            for (int d = 0; d < head_dim; d++) {
                float weighted_sum = 0.0f;
                for (int j = 0; j < seq_len; j++) {
                    const int v_idx = ((b * num_heads + h) * seq_len + j) * head_dim + d;
                    weighted_sum += scores[j] * V[v_idx];
                }
                const int out_idx = ((b * num_heads + h) * seq_len + i) * head_dim + d;
                output[out_idx] = weighted_sum;
            }
        }
    }
} 

// Implementation of MultiHeadAttention::forward_cuda
#ifdef CUDA_AVAILABLE
Matrix MultiHeadAttention::forward_cuda(const Matrix& input, 
                                      const AttentionMask& mask,
                                      const std::optional<KVCache>& kv_cache) {
    const int batch_size = input.rows();
    const int seq_len = input.cols() / hidden_size;
    const float scale = 1.0f / std::sqrt(static_cast<float>(head_dim));
    
    // Project input to Q, K, V using CUDA matmul
    Matrix Q(input.rows(), get_query_weights().cols());
    cuda::matmul(input, get_query_weights(), Q, nullptr);
    
    Matrix K;
    if (kv_cache) {
        K = kv_cache->get_key();
    } else {
        K = Matrix(input.rows(), get_key_weights().cols());
        cuda::matmul(input, get_key_weights(), K, nullptr);
    }
    
    Matrix V;
    if (kv_cache) {
        V = kv_cache->get_value();
    } else {
        V = Matrix(input.rows(), get_value_weights().cols());
        cuda::matmul(input, get_value_weights(), V, nullptr);
    }
    
    // Reshape matrices for attention
    Matrix Q_reshaped = Q.reshape(batch_size, num_heads, seq_len, head_dim);
    Matrix K_reshaped = K.reshape(batch_size, num_heads, seq_len, head_dim);
    Matrix V_reshaped = V.reshape(batch_size, num_heads, seq_len, head_dim);
    
    // Allocate output
    Matrix output(batch_size * seq_len, hidden_size);
    
    // Launch attention kernel
    cuda::launch_attention_kernel(
        Q_reshaped.data(), K_reshaped.data(), V_reshaped.data(),
        output.data(), mask.value().data(),
        batch_size, num_heads, seq_len, head_dim,
        scale);
    
    // Project output
    Matrix final_output(output.rows(), get_output_weights().cols());
    cuda::matmul(output, get_output_weights(), final_output, nullptr);
    
    return final_output;
}
#endif 