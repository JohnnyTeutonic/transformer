#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/tensor_kernels.cuh"

extern hipblasHandle_t cublas_handle;

void launch_tensor_mul(const float* a, const float* b, float* result,
                      int d1, int d2, int d3, int d4, int b_d4) {
    // Treat the tensor multiplication as a batch of matrix multiplications
    // Each batch corresponds to a combination of d1 and d2 indices
    
    const int batch_size = d1 * d2;  // Number of matrix multiplications
    const int m = d3;                // Rows of A
    const int n = b_d4;              // Cols of B
    const int k = d4;                // Cols of A / Rows of B
    
    // Create arrays of pointers for batched operation
    const float** a_array = nullptr;
    const float** b_array = nullptr;
    float** c_array = nullptr;
    
    // Host arrays for storing device pointers
    const float** h_a_array = new const float*[batch_size];
    const float** h_b_array = new const float*[batch_size];
    float** h_c_array = new float*[batch_size];
    
    // Allocate device memory for pointer arrays
    CUDA_CHECK(hipMalloc(&a_array, batch_size * sizeof(float*)));
    CUDA_CHECK(hipMalloc(&b_array, batch_size * sizeof(float*)));
    CUDA_CHECK(hipMalloc(&c_array, batch_size * sizeof(float*)));
    
    // Set up pointers for each batch
    for (int i = 0; i < d1; ++i) {
        for (int j = 0; j < d2; ++j) {
            const int batch_idx = i * d2 + j;
            const size_t offset = (i * d2 * d3 * d4 + j * d3 * d4);
            
            h_a_array[batch_idx] = a + offset;
            h_b_array[batch_idx] = b + offset;
            h_c_array[batch_idx] = result + (i * d2 * d3 * b_d4 + j * d3 * b_d4);
        }
    }
    
    // Copy pointer arrays to device
    CUDA_CHECK(hipMemcpy(a_array, h_a_array, batch_size * sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(b_array, h_b_array, batch_size * sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(c_array, h_c_array, batch_size * sizeof(float*), hipMemcpyHostToDevice));
    
    // Perform batched matrix multiplication
    float alpha = 1.0f;
    float beta = 0.0f;
    
    CUBLAS_CHECK(hipblasSgemmBatched(cublas_handle,
                                   HIPBLAS_OP_N, HIPBLAS_OP_N,
                                   n, m, k,
                                   &alpha,
                                   b_array, n,    // Leading dimension of each B
                                   a_array, k,    // Leading dimension of each A
                                   &beta,
                                   c_array, n,    // Leading dimension of each C
                                   batch_size));  // Number of batches
    
    // Cleanup
    CUDA_CHECK(hipFree(a_array));
    CUDA_CHECK(hipFree(b_array));
    CUDA_CHECK(hipFree(c_array));
    
    delete[] h_a_array;
    delete[] h_b_array;
    delete[] h_c_array;
} 