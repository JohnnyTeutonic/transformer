#include "cuda/cuda_utils.cuh"

hipblasHandle_t cublas_handle;

void initialize_cuda() {
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
}

void cleanup_cuda() {
    if (cublas_handle != nullptr) {
        CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    }
}