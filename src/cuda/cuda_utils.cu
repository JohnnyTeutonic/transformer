#include "hip/hip_runtime.h"
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

namespace cuda {

static bool cuda_initialized = false;
static hipStream_t compute_stream = nullptr;
static hipblasHandle_t cublas_handle = nullptr;

bool is_available() {
    int device_count;
    hipError_t error = hipGetDeviceCount(&device_count);
    return (error == hipSuccess && device_count > 0 && cuda_initialized);
}

void initialize() {
    if (cuda_initialized) return;

    try {
        // Set device
        CUDA_CHECK(hipSetDevice(0));

        // Create stream
        CUDA_CHECK(hipStreamCreate(&compute_stream));

        // Initialize cuBLAS
        CUBLAS_CHECK(hipblasCreate(&cublas_handle));
        CUBLAS_CHECK(hipblasSetStream(cublas_handle, compute_stream));

        cuda_initialized = true;
        std::cout << "CUDA initialized successfully" << std::endl;
    } catch (const std::exception& e) {
        std::cerr << "CUDA initialization failed: " << e.what() << std::endl;
        cleanup();
        throw;
    }
}

void cleanup() {
    if (cublas_handle) {
        hipblasDestroy(cublas_handle);
        cublas_handle = nullptr;
    }
    if (compute_stream) {
        hipStreamDestroy(compute_stream);
        compute_stream = nullptr;
    }
    cuda_initialized = false;
}

hipStream_t get_stream() {
    return compute_stream;
}

hipblasHandle_t get_cublas_handle() {
    return cublas_handle;
}

void synchronize() {
    if (compute_stream) {
        CUDA_CHECK(hipStreamSynchronize(compute_stream));
    }
}

template<typename T>
T* device_malloc(size_t size) {
    T* ptr;
    CUDA_CHECK(hipMalloc(&ptr, size * sizeof(T)));
    return ptr;
}

template<typename T>
void device_free(T* ptr) {
    if (ptr) {
        CUDA_CHECK(hipFree(ptr));
    }
}

template<typename T>
void copy_to_device(T* dst, const T* src, size_t size) {
    CUDA_CHECK(hipMemcpyAsync(dst, src, size * sizeof(T), 
                              hipMemcpyHostToDevice, compute_stream));
}

template<typename T>
void copy_to_host(T* dst, const T* src, size_t size) {
    CUDA_CHECK(hipMemcpyAsync(dst, src, size * sizeof(T), 
                              hipMemcpyDeviceToHost, compute_stream));
}

// Explicit template instantiations
template float* device_malloc<float>(size_t);
template void device_free<float>(float*);
template void copy_to_device<float>(float*, const float*, size_t);
template void copy_to_host<float>(float*, const float*, size_t);

__global__ void softmax_kernel(float* scores, int seq_len) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < seq_len) {
        float max_val = scores[row * seq_len];
        for (int i = 1; i < seq_len; i++) {
            max_val = max(max_val, scores[row * seq_len + i]);
        }

        float sum = 0.0f;
        for (int i = 0; i < seq_len; i++) {
            scores[row * seq_len + i] = expf(scores[row * seq_len + i] - max_val);
            sum += scores[row * seq_len + i];
        }

        for (int i = 0; i < seq_len; i++) {
            scores[row * seq_len + i] /= sum;
        }
    }
}

void launch_softmax_kernel(float* scores, int seq_len, hipStream_t stream) {
    dim3 block_dim(256);
    dim3 grid_dim((seq_len + block_dim.x - 1) / block_dim.x);

    softmax_kernel<<<grid_dim, block_dim, 0, stream>>>(scores, seq_len);
}

Matrix cuda_matmul(const Matrix& A, const Matrix& B) {
    std::cout << "Starting CUDA matrix multiplication..." << std::endl;
    std::cout << "Matrix A: " << A.rows() << "x" << A.cols() << std::endl;
    std::cout << "Matrix B: " << B.rows() << "x" << B.cols() << std::endl;

    hipblasHandle_t handle;
    hipblasStatus_t status;
    hipError_t err;

    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("Failed to create cuBLAS handle");
    }

    float alpha = 1.0f;
    float beta = 0.0f;

    Matrix C(A.rows(), B.cols(), 0.0f);
    Matrix C_gpu = C.to_gpu();
    std::cout << "Created output matrix C: " << C.rows() << "x" << C.cols() << std::endl;

    try {
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, B.cols(), A.rows(), A.cols(), &alpha,
                             B.get_data(), B.cols(), A.get_data(), A.cols(), &beta,
                             C_gpu.get_data(), C_gpu.cols());

        if (status != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("cuBLAS SGEMM failed with status: " + std::to_string(status));
        }

        // Synchronize to catch any asynchronous errors
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            throw std::runtime_error("CUDA sync failed: " + std::string(hipGetErrorString(err)));
        }

        std::cout << "CUDA matrix multiplication completed successfully" << std::endl;
        C = C_gpu.to_cpu();
    } catch (const std::exception& e) {
        hipblasDestroy(handle);
        throw;
    }

    hipblasDestroy(handle);
    return C;
}

} // namespace cuda