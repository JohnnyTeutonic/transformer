#include "hip/hip_runtime.h"
#include "../../include/attention.hpp"
#include "../../include/cuda/matrix_ops.cuh"
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/attention_ops.cuh"
#include <hip/hip_runtime.h>

Matrix MultiHeadAttention::forward_cuda(const Matrix& input, 
                                      const AttentionMask& mask,
                                      const std::optional<KVCache>& kv_cache) {
    const int batch_size = input.rows();
    const int seq_len = input.cols() / hidden_size_;
    const float scale = 1.0f / std::sqrt(static_cast<float>(head_dim_));
    
    // Project input to Q, K, V spaces
    Matrix Q(input.rows(), params_.query_weights.cols());
    cuda::matmul(input, params_.query_weights, Q, nullptr);
    
    Matrix K;
    if (kv_cache) {
        K = kv_cache->get_key();
    } else {
        K = Matrix(input.rows(), params_.key_weights.cols());
        cuda::matmul(input, params_.key_weights, K, nullptr);
    }
    
    Matrix V;
    if (kv_cache) {
        V = kv_cache->get_value();
    } else {
        V = Matrix(input.rows(), params_.value_weights.cols());
        cuda::matmul(input, params_.value_weights, V, nullptr);
    }
    
    // Allocate output
    Matrix output(batch_size * seq_len, hidden_size_);
    
    // Launch attention kernel
    cuda::launch_attention_kernel(
        Q.data(), K.data(), V.data(),
        output.data(), mask.value().data(),
        static_cast<int>(batch_size), 
        static_cast<int>(num_heads_), 
        static_cast<int>(seq_len), 
        static_cast<int>(head_dim_),
        scale, cuda::get_stream());
    
    // Project output
    Matrix output_proj(output.rows(), params_.output_weights.cols());
    cuda::matmul(output, params_.output_weights, output_proj, nullptr);
    return output_proj;
} 