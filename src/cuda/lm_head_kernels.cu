#include "hip/hip_runtime.h"
#include "../../include/lm_head.hpp"
#include "../../include/cuda/lm_head_kernels.cuh"
#include "../../include/cuda/cuda_check.cuh"

#if defined(USE_CUDA) && defined(CUDA_AVAILABLE)
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace {
// Anonymous namespace for kernel definitions
__global__ void convert_to_fp16_kernel_impl(half* output, const float* input, size_t size) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(input[idx]);
    }
}

__global__ void convert_and_expand_vocab_kernel_impl(
    float* output, const half* input, const unsigned char* active_tokens,
    size_t batch_size, size_t vocab_size, size_t active_vocab_size) {
    
    const size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < batch_size && col < vocab_size) {
        // Find position in compressed vocabulary
        size_t active_pos = 0;
        for (size_t i = 0; i < col; i++) {
            if (active_tokens[i]) {
                active_pos++;
            }
        }
        
        if (active_tokens[col]) {
            output[row * vocab_size + col] = __half2float(input[row * active_vocab_size + active_pos]);
        } else {
            output[row * vocab_size + col] = -INFINITY;
        }
    }
}
} // anonymous namespace

// Device function implementations
__device__ void LanguageModelHead::convert_to_fp16_kernel(
    half* output, const float* input, size_t idx) {
    output[idx] = __float2half(input[idx]);
}

__device__ void LanguageModelHead::convert_and_expand_vocab_kernel(
    float* output, const half* input, const unsigned char* active_tokens,
    size_t row, size_t col, size_t batch_size, size_t vocab_size, size_t active_vocab_size) {
    
    if (row < batch_size && col < vocab_size) {
        size_t active_pos = 0;
        for (size_t i = 0; i < col; i++) {
            if (active_tokens[i]) {
                active_pos++;
            }
        }
        
        if (active_tokens[col]) {
            output[row * vocab_size + col] = __half2float(input[row * active_vocab_size + active_pos]);
        } else {
            output[row * vocab_size + col] = -INFINITY;
        }
    }
}

// Host function implementations
void LanguageModelHead::launch_convert_to_fp16(half* output, const float* input, size_t size) {
    const int block_size = 256;
    const int num_blocks = (size + block_size - 1) / block_size;
    convert_to_fp16_kernel_impl<<<num_blocks, block_size, 0, compute_stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void LanguageModelHead::launch_convert_and_expand_vocab(
    float* output, const half* input,
    size_t batch_size, size_t vocab_size, size_t active_vocab_size) {
    
    dim3 block_dim(16, 16);
    dim3 grid_dim(
        (batch_size + block_dim.x - 1) / block_dim.x,
        (vocab_size + block_dim.y - 1) / block_dim.y
    );
    
    convert_and_expand_vocab_kernel_impl<<<grid_dim, block_dim, 0, compute_stream>>>(
        output, input, active_tokens.data(),
        batch_size, vocab_size, active_vocab_size
    );
    CUDA_CHECK(hipGetLastError());
}

namespace cuda {
    // Remove duplicate utility functions and keep only the kernel-specific code
    
    void launch_add_bias(float* output, const float* bias, int rows, int cols) {
        dim3 block(256);
        dim3 grid((rows * cols + block.x - 1) / block.x);
        add_bias_kernel<<<grid, block>>>(output, bias, rows, cols);
    }

    void launch_row_sum(const float* input, float* output, int rows, int cols) {
        dim3 block(256);
        dim3 grid((cols + block.x - 1) / block.x);
        row_sum_kernel<<<grid, block>>>(input, output, rows, cols);
    }

    void launch_adam_update(float* params, const float* grads, float* m, float* v,
                          float beta1, float beta2, float eps, float lr, int size,
                          unsigned long step, hipStream_t stream) {
        dim3 block(256);
        dim3 grid((size + block.x - 1) / block.x);
        adam_update_kernel<<<grid, block, 0, stream>>>(params, grads, m, v,
                                                      beta1, beta2, eps, lr, size, step);
    }
}

#endif // defined(USE_CUDA) && defined(CUDA_AVAILABLE) 