#include "hip/hip_runtime.h"
#include "../../include/lm_head.hpp"
#include "../../include/cuda/lm_head_kernels.cuh"
#include "../../include/cuda/cuda_check.cuh"

#if defined(USE_CUDA) && defined(CUDA_AVAILABLE)
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace {
// Anonymous namespace for kernel definitions
__global__ void convert_to_fp16_kernel_impl(half* output, const float* input, size_t size) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(input[idx]);
    }
}

__global__ void convert_and_expand_vocab_kernel_impl(
    float* output, const half* input, const unsigned char* active_tokens,
    size_t batch_size, size_t vocab_size, size_t active_vocab_size) {
    
    const size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < batch_size && col < vocab_size) {
        // Find position in compressed vocabulary
        size_t active_pos = 0;
        for (size_t i = 0; i < col; i++) {
            if (active_tokens[i]) {
                active_pos++;
            }
        }
        
        if (active_tokens[col]) {
            output[row * vocab_size + col] = __half2float(input[row * active_vocab_size + active_pos]);
        } else {
            output[row * vocab_size + col] = -INFINITY;
        }
    }
}
} // anonymous namespace

// Device function implementations
__device__ void LanguageModelHead::convert_to_fp16_kernel(
    half* output, const float* input, size_t idx) {
    output[idx] = __float2half(input[idx]);
}

__device__ void LanguageModelHead::convert_and_expand_vocab_kernel(
    float* output, const half* input, const unsigned char* active_tokens,
    size_t row, size_t col, size_t batch_size, size_t vocab_size, size_t active_vocab_size) {
    
    if (row < batch_size && col < vocab_size) {
        size_t active_pos = 0;
        for (size_t i = 0; i < col; i++) {
            if (active_tokens[i]) {
                active_pos++;
            }
        }
        
        if (active_tokens[col]) {
            output[row * vocab_size + col] = __half2float(input[row * active_vocab_size + active_pos]);
        } else {
            output[row * vocab_size + col] = -INFINITY;
        }
    }
}

// Host function implementations
void LanguageModelHead::launch_convert_to_fp16(half* output, const float* input, size_t size) {
    const int block_size = 256;
    const int num_blocks = (size + block_size - 1) / block_size;
    convert_to_fp16_kernel_impl<<<num_blocks, block_size, 0, compute_stream>>>(output, input, size);
    CUDA_CHECK(hipGetLastError());
}

void LanguageModelHead::launch_convert_and_expand_vocab(
    float* output, const half* input,
    size_t batch_size, size_t vocab_size, size_t active_vocab_size) {
    
    dim3 block_dim(16, 16);
    dim3 grid_dim(
        (batch_size + block_dim.x - 1) / block_dim.x,
        (vocab_size + block_dim.y - 1) / block_dim.y
    );
    
    convert_and_expand_vocab_kernel_impl<<<grid_dim, block_dim, 0, compute_stream>>>(
        output, input, active_tokens.data(),
        batch_size, vocab_size, active_vocab_size
    );
    CUDA_CHECK(hipGetLastError());
}

namespace cuda {

// Forward declarations of kernels
__global__ void add_bias_kernel(float* output, const float* bias, int rows, int cols);
__global__ void row_sum_kernel(const float* input, float* output, int rows, int cols);
__global__ void adam_update_kernel(float* params, const float* grads, float* m, float* v,
                                 float beta1, float beta2, float lr, float epsilon, int size);

bool is_available() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return (error == hipSuccess && deviceCount > 0);
}

hipStream_t get_stream() {
    return nullptr; // Returns default stream for now
}

void synchronize() {
    CUDA_CHECK(hipDeviceSynchronize());
}

void launch_add_bias(float* output, const float* bias, int rows, int cols) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_bias_kernel<<<grid, block>>>(output, bias, rows, cols);
}

void launch_row_sum(const float* input, float* output, int rows, int cols) {
    dim3 block(256);
    dim3 grid((cols + block.x - 1) / block.x);
    row_sum_kernel<<<grid, block>>>(input, output, rows, cols);
}

void launch_adam_update(float* params, const float* grads, float* m, float* v,
                      float beta1, float beta2, float lr, float epsilon, int size,
                      hipStream_t stream) {
    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);
    adam_update_kernel<<<grid, block, 0, stream>>>(params, grads, m, v,
                                                  beta1, beta2, lr, epsilon, size);
}

// Kernel implementations
__global__ void add_bias_kernel(float* output, const float* bias, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        int col = idx % cols;
        output[idx] += bias[col];
    }
}

__global__ void row_sum_kernel(const float* input, float* output, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < cols) {
        float sum = 0.0f;
        for (int row = 0; row < rows; row++) {
            sum += input[row * cols + col];
        }
        output[col] = sum;
    }
}

__global__ void adam_update_kernel(float* params, const float* grads, float* m, float* v,
                                 float beta1, float beta2, float lr, float epsilon, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Update biased first moment estimate
        m[idx] = beta1 * m[idx] + (1.0f - beta1) * grads[idx];
        
        // Update biased second raw moment estimate
        v[idx] = beta2 * v[idx] + (1.0f - beta2) * grads[idx] * grads[idx];
        
        // Update parameters
        params[idx] -= lr * m[idx] / (sqrtf(v[idx]) + epsilon);
    }
}

} // namespace cuda 

#endif // defined(USE_CUDA) && defined(CUDA_AVAILABLE) 