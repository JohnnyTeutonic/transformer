#include "hip/hip_runtime.h"
#include "../../include/cuda/feed_forward_kernels.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/matrix_ops.cuh"  // Include this for launch_add_bias
#include "../../include/matrix.hpp"
#include "../../include/feed_forward.hpp"
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/kernel_declarations.cuh"
#include <hip/hip_runtime.h>

// Forward declare kernels
namespace cuda {
    __global__ void add_bias_kernel(float* output, const float* bias,
                                  int batch_size, int hidden_size);
    __global__ void gelu_activation_kernel(float* data, int size);
}

namespace cuda {
    __global__ void feed_forward_backward_kernel_1(const float* grad, const float* w2,
                                                 float* d_intermediate, int batch_size,
                                                 int hidden_size, int intermediate_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int total_elements = batch_size * intermediate_size;
        
        if (idx < total_elements) {
            int batch = idx / intermediate_size;
            int inter = idx % intermediate_size;
            
            float sum = 0.0f;
            for (int k = 0; k < hidden_size; ++k) {
                sum += grad[batch * hidden_size + k] * w2[inter * hidden_size + k];
            }
            d_intermediate[idx] = sum;
        }
    }

    __global__ void feed_forward_backward_kernel_2(const float* d_intermediate, const float* w1,
                                                 float* dx, int batch_size,
                                                 int hidden_size, int intermediate_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int total_elements = batch_size * hidden_size;
        
        if (idx < total_elements) {
            int batch = idx / hidden_size;
            int hidden = idx % hidden_size;
            
            float sum = 0.0f;
            for (int k = 0; k < intermediate_size; ++k) {
                sum += d_intermediate[batch * intermediate_size + k] * w1[hidden * intermediate_size + k];
            }
            dx[idx] = sum;
        }
    }

    void feed_forward_backward(const Matrix& grad, const Matrix& weights, 
                             Matrix& dx, bool is_first_layer) {
        const int batch_size = grad.rows();
        const int hidden_size = weights.cols();
        const int intermediate_size = weights.rows();
        
        float* d_grad, *d_weights, *d_dx, *d_intermediate;
        size_t grad_size = grad.size() * sizeof(float);
        size_t weights_size = weights.size() * sizeof(float);
        size_t dx_size = dx.size() * sizeof(float);
        size_t intermediate_size_bytes = batch_size * intermediate_size * sizeof(float);
        
        CUDA_CHECK(hipMalloc(&d_grad, grad_size));
        CUDA_CHECK(hipMalloc(&d_weights, weights_size));
        CUDA_CHECK(hipMalloc(&d_dx, dx_size));
        CUDA_CHECK(hipMalloc(&d_intermediate, intermediate_size_bytes));
        
        CUDA_CHECK(hipMemcpy(d_grad, grad.data(), grad_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_weights, weights.data(), weights_size, hipMemcpyHostToDevice));
        
        dim3 block(256);
        dim3 grid((batch_size * intermediate_size + 255) / 256);
        
        feed_forward_backward_kernel_1<<<grid, block>>>(
            d_grad, d_weights, d_intermediate, batch_size, hidden_size, intermediate_size);
            
        if (!is_first_layer) {
            gelu_backward_kernel<<<grid, block>>>(
                d_intermediate, d_dx, batch_size * intermediate_size);
                
            feed_forward_backward_kernel_2<<<grid, block>>>(
                d_intermediate, d_weights, d_dx, batch_size, hidden_size, intermediate_size);
        }
        
        CUDA_CHECK(hipMemcpy(dx.data(), d_dx, dx_size, hipMemcpyDeviceToHost));
        
        CUDA_CHECK(hipFree(d_grad));
        CUDA_CHECK(hipFree(d_weights));
        CUDA_CHECK(hipFree(d_dx));
        CUDA_CHECK(hipFree(d_intermediate));
    }

    __global__ void gelu_backward_kernel(const float* d_intermediate, float* d_input,
                                       const int num_elements) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < num_elements) {
            float x = d_input[idx];
            float cdf = 0.5f * (1.0f + tanhf(0.797884f * (x + 0.044715f * x * x * x)));
            float pdf = 0.797884f * (1.0f - tanhf(0.797884f * x) * tanhf(0.797884f * x));
            d_input[idx] = d_intermediate[idx] * (cdf + x * pdf);
        }
    }

    __global__ void gelu_activation_kernel(float* data, int size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            float x = data[idx];
            float x3 = x * x * x;
            constexpr float sqrt_2_pi = 0.7978845608028654f;
            data[idx] = 0.5f * x * (1.0f + tanhf(sqrt_2_pi * (x + 0.044715f * x3)));
        }
    }

    void feed_forward_forward(const Matrix& input, const Matrix& W1, const Matrix& W2,
                            Matrix& intermediate, Matrix& output) {
        // Implementation...
    }
    
    void feed_forward_backward(const Matrix& grad_output, const Matrix& input,
                             const Matrix& W1, const Matrix& W2,
                             Matrix& d_input, Matrix& d_W1, Matrix& d_W2) {
        // Implementation...
    }
} // end namespace cuda

// Move FeedForward::forward_cuda implementation outside of cuda namespace
#ifdef CUDA_AVAILABLE
Matrix FeedForward::forward_cuda(const Matrix& input) {
    const int batch_size = input.rows();
    const int hidden_size = input.cols();
    
    // Get weights and biases using accessor methods
    const Matrix& W1 = get_fc1_weights();
    const Matrix& W2 = get_fc2_weights();
    const Vector& b1 = get_fc1_bias();
    const Vector& b2 = get_fc2_bias();
    
    // First linear layer
    Matrix intermediate(batch_size, W1.cols());  // Create intermediate matrix with correct dimensions
    cuda::matmul(input, W1, intermediate, nullptr);
    
    // Add bias - ensure types match exactly
    cuda::launch_add_bias(intermediate.data(), b1.data(),
                         static_cast<int>(batch_size), 
                         static_cast<int>(intermediate.cols()));
    
    // Apply GELU activation
    dim3 block(256);
    dim3 grid((intermediate.size() + block.x - 1) / block.x);
    
    cuda::gelu_activation_kernel<<<grid, block>>>(
        intermediate.data(),
        static_cast<int>(intermediate.size())
    );
    CUDA_CHECK(hipGetLastError());
    
    // Second linear layer
    Matrix output(batch_size, W2.cols());  // Create output matrix with correct dimensions
    cuda::matmul(intermediate, W2, output, nullptr);
    
    // Add bias - ensure types match exactly
    cuda::launch_add_bias(output.data(), b2.data(),
                         static_cast<int>(batch_size), 
                         static_cast<int>(output.cols()));
    
    return output;
}
#endif