#include "hip/hip_runtime.h"
#include "../../include/cuda/feed_forward_kernels.cuh"
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/matrix_ops.cuh"  // Include this for launch_add_bias
#include "../../include/matrix.hpp"
#include "../../include/feed_forward.hpp"
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/kernel_declarations.cuh"
#include <hip/hip_runtime.h>

// Forward declare kernels
namespace cuda {
    __global__ void add_bias_kernel(float* output, const float* bias,
                                  int batch_size, int hidden_size);
    __global__ void gelu_activation_kernel(float* data, int size);
}

namespace cuda {
    __global__ void feed_forward_backward_kernel_1(const float* grad, const float* w2,
                                                 float* d_intermediate, int batch_size,
                                                 int hidden_size, int intermediate_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int total_elements = batch_size * intermediate_size;
        
        if (idx < total_elements) {
            int batch = idx / intermediate_size;
            int inter = idx % intermediate_size;
            
            float sum = 0.0f;
            for (int k = 0; k < hidden_size; ++k) {
                sum += grad[batch * hidden_size + k] * w2[inter * hidden_size + k];
            }
            d_intermediate[idx] = sum;
        }
    }

    __global__ void feed_forward_backward_kernel_2(const float* d_intermediate, const float* w1,
                                                 float* dx, int batch_size,
                                                 int hidden_size, int intermediate_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int total_elements = batch_size * hidden_size;
        
        if (idx < total_elements) {
            int batch = idx / hidden_size;
            int hidden = idx % hidden_size;
            
            float sum = 0.0f;
            for (int k = 0; k < intermediate_size; ++k) {
                sum += d_intermediate[batch * intermediate_size + k] * w1[hidden * intermediate_size + k];
            }
            dx[idx] = sum;
        }
    }

    void feed_forward_backward(const Matrix& grad, const Matrix& weights, 
                             Matrix& dx, bool is_first_layer) {
        const int batch_size = grad.rows();
        const int hidden_size = weights.cols();
        const int intermediate_size = weights.rows();
        
        float* d_grad, *d_weights, *d_dx, *d_intermediate;
        size_t grad_size = grad.size() * sizeof(float);
        size_t weights_size = weights.size() * sizeof(float);
        size_t dx_size = dx.size() * sizeof(float);
        size_t intermediate_size_bytes = batch_size * intermediate_size * sizeof(float);
        
        CUDA_CHECK(hipMalloc(&d_grad, grad_size));
        CUDA_CHECK(hipMalloc(&d_weights, weights_size));
        CUDA_CHECK(hipMalloc(&d_dx, dx_size));
        CUDA_CHECK(hipMalloc(&d_intermediate, intermediate_size_bytes));
        
        CUDA_CHECK(hipMemcpy(d_grad, grad.data(), grad_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_weights, weights.data(), weights_size, hipMemcpyHostToDevice));
        
        dim3 block(256);
        dim3 grid((batch_size * intermediate_size + 255) / 256);
        
        feed_forward_backward_kernel_1<<<grid, block>>>(
            d_grad, d_weights, d_intermediate, batch_size, hidden_size, intermediate_size);
            
        if (!is_first_layer) {
            gelu_backward_kernel<<<grid, block>>>(
                d_intermediate, d_dx, batch_size * intermediate_size);
                
            feed_forward_backward_kernel_2<<<grid, block>>>(
                d_intermediate, d_weights, d_dx, batch_size, hidden_size, intermediate_size);
        }
        
        CUDA_CHECK(hipMemcpy(dx.data(), d_dx, dx_size, hipMemcpyDeviceToHost));
        
        CUDA_CHECK(hipFree(d_grad));
        CUDA_CHECK(hipFree(d_weights));
        CUDA_CHECK(hipFree(d_dx));
        CUDA_CHECK(hipFree(d_intermediate));
    }

    __global__ void gelu_backward_kernel(const float* d_intermediate, float* d_input,
                                       const int num_elements) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < num_elements) {
            float x = d_input[idx];
            float cdf = 0.5f * (1.0f + tanhf(0.797884f * (x + 0.044715f * x * x * x)));
            float pdf = 0.797884f * (1.0f - tanhf(0.797884f * x) * tanhf(0.797884f * x));
            d_input[idx] = d_intermediate[idx] * (cdf + x * pdf);
        }
    }

    __global__ void gelu_activation_kernel(float* data, int size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size) {
            float x = data[idx];
            float x3 = x * x * x;
            constexpr float sqrt_2_pi = 0.7978845608028654f;
            data[idx] = 0.5f * x * (1.0f + tanhf(sqrt_2_pi * (x + 0.044715f * x3)));
        }
    }

    void feed_forward_forward(const Matrix& input, const Matrix& W1, const Matrix& W2,
                            Matrix& intermediate, Matrix& output) {
        // Implementation...
    }
    
    void feed_forward_backward(const Matrix& grad_output, const Matrix& input,
                             const Matrix& W1, const Matrix& W2,
                             Matrix& d_input, Matrix& d_W1, Matrix& d_W2) {
        // Implementation...
    }
} // end namespace cuda

// Move FeedForward::forward_cuda implementation outside of cuda namespace
#ifdef CUDA_AVAILABLE
Matrix FeedForward::forward_cuda(const Matrix& input) {
    const int batch_size = input.rows();
    const int hidden_size = input.cols();
    
    // Get weights and biases using accessor methods
    const Matrix& W1 = get_fc1_weights();
    const Matrix& W2 = get_fc2_weights();
    const Vector& b1 = get_fc1_bias();
    const Vector& b2 = get_fc2_bias();
    
    // First linear layer
    Matrix intermediate(batch_size, W1.cols());  // Create intermediate matrix with correct dimensions
    
    // Allocate GPU memory for intermediate results and biases
    float* d_intermediate;
    float* d_b1;
    size_t intermediate_size = intermediate.size() * sizeof(float);
    size_t b1_size = b1.size() * sizeof(float);
    
    CUDA_CHECK(hipMalloc(&d_intermediate, intermediate_size));
    CUDA_CHECK(hipMalloc(&d_b1, b1_size));
    
    // Copy bias to GPU
    CUDA_CHECK(hipMemcpy(d_b1, b1.data(), b1_size, hipMemcpyHostToDevice));
    
    // First matmul (this already handles GPU memory internally)
    cuda::matmul(input, W1, intermediate, nullptr);
    
    // Copy intermediate result to GPU
    CUDA_CHECK(hipMemcpy(d_intermediate, intermediate.data(), intermediate_size, hipMemcpyHostToDevice));
    
    // Add bias using GPU memory pointers
    cuda::launch_add_bias(d_intermediate, d_b1,
                         static_cast<int>(batch_size), 
                         static_cast<int>(intermediate.cols()));
    
    // Apply GELU activation
    dim3 block(256);
    dim3 grid((intermediate.size() + block.x - 1) / block.x);
    
    cuda::gelu_activation_kernel<<<grid, block>>>(
        d_intermediate,  // Use GPU pointer
        static_cast<int>(intermediate.size())
    );
    CUDA_CHECK(hipGetLastError());
    
    // Copy result back to intermediate
    CUDA_CHECK(hipMemcpy(intermediate.data(), d_intermediate, intermediate_size, hipMemcpyDeviceToHost));
    
    // Second linear layer
    Matrix output(batch_size, W2.cols());
    
    // Allocate GPU memory for output and second bias
    float* d_output;
    float* d_b2;
    size_t output_size = output.size() * sizeof(float);
    size_t b2_size = b2.size() * sizeof(float);
    
    CUDA_CHECK(hipMalloc(&d_output, output_size));
    CUDA_CHECK(hipMalloc(&d_b2, b2_size));
    
    // Copy second bias to GPU
    CUDA_CHECK(hipMemcpy(d_b2, b2.data(), b2_size, hipMemcpyHostToDevice));
    
    // Second matmul
    cuda::matmul(intermediate, W2, output, nullptr);
    
    // Copy output to GPU
    CUDA_CHECK(hipMemcpy(d_output, output.data(), output_size, hipMemcpyHostToDevice));
    
    // Add second bias
    cuda::launch_add_bias(d_output, d_b2,
                         static_cast<int>(batch_size), 
                         static_cast<int>(output.cols()));
    
    // Copy final result back to output
    CUDA_CHECK(hipMemcpy(output.data(), d_output, output_size, hipMemcpyDeviceToHost));
    
    // Cleanup GPU memory
    CUDA_CHECK(hipFree(d_intermediate));
    CUDA_CHECK(hipFree(d_b1));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_b2));
    
    return output;
}
#endif